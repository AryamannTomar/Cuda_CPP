#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include <stdlib.h>

/*
    Main Fn will launch 1 Dimensional Grid with 1 thread Block which have 8 threads.
*/

__global__ void unique_idx_calc_threadIdx(int *input){
    int tid=threadIdx.x;
    printf("threadIdx: %d, Value: %d \n", tid, input[tid]);
}

__global__ void unique_gid_calculation(int *input){
    int tid=threadIdx.x;
    int offset=blockIdx.x*blockDim.x;
    int gid=tid+offset;
    printf("blockIdx: %d, threadIdx: %d, gid, %d, Value: %d \n", blockIdx.x, tid, gid, input[tid]);
}

int main(){

    /*
      int array_size=16;
      int array_byte_size=sizeof(int)*array_size;
      int h_data[]={23,9,4,53,65,12,1,33,87,45,23,12,342,56,44,99};  

      dim3 block(4);
      dim3 grid(4);

      unique_gid_calculation << <grid, block> >> (d_data);
    */

    int array_size=8;
    int array_byte_size=sizeof(int)*array_size;
    int h_data[]={23,9,4,53,65,12,1,33};
    for (int i=0; i<array_size;i++){
        printf("%d", h_data[i]);
    }
    printf("\n \n");
    
    int *d_data;
    hipMalloc((void**)&d_data, array_byte_size);
    hipMemcpy(d_data,h_data,array_byte_size,hipMemcpyHostToDevice);

    dim3 block(8);
    dim3 grid(1);
    unique_idx_calc_threadIdx << <grid,block> >>(d_data);
    hipDeviceSynchronize(); 
    hipDeviceReset();
    return 0;
}