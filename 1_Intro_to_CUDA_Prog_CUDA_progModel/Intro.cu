#include "hip/hip_runtime.h"

#include "stdio.h"

__global__ void hello_cuda(){
    printf("Hello CUDA World! \n");
}

int main(){
    /*
        Grid: Collection of All threads launch for a kernel
        Block: Threads in a grid is organized into groups called thread blocks
        
        Kernel Launch Parameters (Total: 4)
        Kernel_Name <<< grid, block >>>();
        Kernel_Name <<< number_of_blocks, thread_per_block >>>(arguments);
    */
    
    /*
        hello_cuda<< <1,1> >> (); 

        Now 20 Threads are going to execute print function
        hello_cuda<< <1,20> >> (); 
    */

    /*
        Implicitly Intialized Variables 
            - threadIdx
            - blockIdx
            - blockDim
            - gridDim

        int nx, ny;
        nx=16;
        ny=4;
        dim3 block(8,2);
        dim3 grid(nx/block.x,ny/block.y);
        hello_cuda<< <grid, block> >>();
    */
    // 32 Thread Blocks arranged into 8 thread blocks ion X dimension each having 4 threads
    dim3 block(4);
    dim3 grid(8);
    hello_cuda<<<grid,block>>>();

    // Gurantees Host Code to wait untill all previous launch kernels finish their execution
    hipDeviceSynchronize(); // As kernel Launch is an asynchronous fn call, Host does not have to wait untill the Kernel Execution Finish
    hipDeviceReset();
    return 0;
}